#include "hip/hip_runtime.h"
/* XMRig
* Copyright 2010      Jeff Garzik <jgarzik@pobox.com>
* Copyright 2012-2014 pooler      <pooler@litecoinpool.org>
* Copyright 2014      Lucas Jones <https://github.com/lucasjones>
* Copyright 2014-2016 Wolf9466    <https://github.com/OhGodAPet>
* Copyright 2016      Jay D Dee   <jayddee246@gmail.com>
* Copyright 2017-2018 XMR-Stak    <https://github.com/fireice-uk>, <https://github.com/psychocrypt>
* Copyright 2018      Lee Clagett <https://github.com/vtnerd>
* Copyright 2016-2018 XMRig       <https://github.com/xmrig>, <support@xmrig.com>
*
*   This program is free software: you can redistribute it and/or modify
*   it under the terms of the GNU General Public License as published by
*   the Free Software Foundation, either version 3 of the License, or
*   (at your option) any later version.
*
*   This program is distributed in the hope that it will be useful,
*   but WITHOUT ANY WARRANTY; without even the implied warranty of
*   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
*   GNU General Public License for more details.
*
*   You should have received a copy of the GNU General Public License
*   along with this program. If not, see <http://www.gnu.org/licenses/>.
*/


#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifdef _WIN32
#include <windows.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
    if (waitTime > 0)
    {
        if (waitTime > 100)
        {
            // use a waitable timer for larger intervals > 0.1ms

            HANDLE timer;
            LARGE_INTEGER ft;

            ft.QuadPart = -10ll * int64_t(waitTime); // Convert to 100 nanosecond interval, negative value indicates relative time

            timer = CreateWaitableTimer(NULL, TRUE, NULL);
            SetWaitableTimer(timer, &ft, 0, NULL, NULL, 0);
            WaitForSingleObject(timer, INFINITE);
            CloseHandle(timer);
        }
        else
        {
            // use a polling loop for short intervals <= 100ms

            LARGE_INTEGER perfCnt, start, now;
            __int64 elapsed;

            QueryPerformanceFrequency(&perfCnt);
            QueryPerformanceCounter(&start);
            do {
        SwitchToThread();
                QueryPerformanceCounter((LARGE_INTEGER*) &now);
                elapsed = (__int64)((now.QuadPart - start.QuadPart) / (float)perfCnt.QuadPart * 1000 * 1000);
            } while ( elapsed < waitTime );
        }
    }
}
#else
#include <unistd.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
    usleep(waitTime);
}
#endif

#include "cryptonight.h"
#include "cuda_extra.h"
#include "cuda_aes.hpp"
#include "cuda_device.hpp"

#if defined(__x86_64__) || defined(_M_AMD64) || defined(__LP64__)
#   define _ASM_PTR_ "l"
#else
#   define _ASM_PTR_ "r"
#endif

/* sm_2X is limited to 2GB due to the small TLB
 * therefore we never use 64bit indices
 */
#if defined(XMR_STAK_LARGEGRID) && (__CUDA_ARCH__ >= 300)
typedef uint64_t IndexType;
#else
typedef int IndexType;
#endif

__device__ __forceinline__ uint64_t cuda_mul128( uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi )
{
    *product_hi = __umul64hi( multiplier, multiplicand );
    return (multiplier * multiplicand );
}

template< typename T >
__device__ __forceinline__ T loadGlobal64( T * const addr )
{
    T x;
    asm volatile( "ld.global.cg.u64 %0, [%1];" : "=l"( x ) : _ASM_PTR_(addr));
    return x;
}

template< typename T >
__device__ __forceinline__ T loadGlobal32( T * const addr )
{
    T x;
    asm volatile( "ld.global.cg.u32 %0, [%1];" : "=r"( x ) : _ASM_PTR_(addr));
    return x;
}


template< typename T >
__device__ __forceinline__ void storeGlobal32( T* addr, T const & val )
{
    asm volatile( "st.global.cg.u32 [%0], %1;" : : _ASM_PTR_(addr), "r"( val ) );
}

template<size_t ITERATIONS, size_t OFFSET>
__global__ void cryptonight_core_gpu_phase1( int threads, int bfactor, int partidx, uint32_t * __restrict__ long_state, uint32_t * __restrict__ ctx_state, uint32_t * __restrict__ ctx_key1 )
{
    __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init( sharedMemory );
    __syncthreads( );

    const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
    const int sub = ( threadIdx.x & 7 ) << 2;

    const int batchsize = ITERATIONS >> bfactor;
    const int start = partidx * batchsize;
    const int end = start + batchsize;

    if ( thread >= threads )
        return;

    uint32_t key[40], text[4];

    MEMCPY8( key, ctx_key1 + thread * 40, 20 );

    if( partidx == 0 )
    {
        // first round
        MEMCPY8( text, ctx_state + thread * 50 + sub + 16, 2 );
    }
    else
    {
        // load previous text data
        MEMCPY8( text, &long_state[( (uint64_t) thread << OFFSET) + sub + start - 32], 2 );
    }
    __syncthreads( );
    for ( int i = start; i < end; i += 32 )
    {
        cn_aes_pseudo_round_mut( sharedMemory, text, key );
        MEMCPY8(&long_state[((uint64_t) thread << OFFSET) + (sub + i)], text, 2);
    }
}

/** avoid warning `unused parameter` */
template< typename T >
__forceinline__ __device__ void unusedVar( const T& )
{
}

/** shuffle data for
 *
 * - this method can be used with all compute architectures
 * - for <sm_30 shared memory is needed
 *
 * @param ptr pointer to shared memory, size must be `threadIdx.x * sizeof(uint32_t)`
 *            value can be NULL for compute architecture >=sm_30
 * @param sub thread number within the group, range [0;4)
 * @param value value to share with other threads within the group
 * @param src thread number within the group from where the data is read, range [0;4)
 */
__forceinline__ __device__ uint32_t shuffle(volatile uint32_t* ptr,const uint32_t sub,const int val,const uint32_t src)
{
#   if ( __CUDA_ARCH__ < 300 )
    ptr[sub] = val;
    return ptr[src&3];
#   else
    unusedVar( ptr );
    unusedVar( sub );

#   if (__CUDACC_VER_MAJOR__ >= 9)
    return __shfl_sync(0xFFFFFFFF, val, src, 4);
#   else
    return __shfl(val, src, 4);
#   endif

#   endif
}

__device__ __forceinline__ uint32_t variant1_1(const uint32_t src)
{
    const uint8_t tmp = src >> 24;
    const uint32_t table = 0x75310;
    const uint8_t index = (((tmp >> 3) & 6) | (tmp & 1)) << 1;
    return (src & 0x00ffffff) | ((tmp ^ ((table >> index) & 0x30)) << 24);
}

template<size_t ITERATIONS, size_t OFFSET, size_t MASK, uint8_t VARIANT>
#ifdef XMR_STAK_THREADS
__launch_bounds__( XMR_STAK_THREADS * 4 )
#endif
__global__ void cryptonight_core_gpu_phase2( int threads, int bfactor, int partidx, uint32_t * d_long_state, uint32_t * d_ctx_a, uint32_t * d_ctx_b, const uint32_t * d_tweak1_2)
{
    __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init( sharedMemory );

    __syncthreads( );

    const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 2;
    const int sub = threadIdx.x & 3;
    const int sub2 = sub & 2;

#if( __CUDA_ARCH__ < 300 )
        extern __shared__ uint32_t shuffleMem[];
        volatile uint32_t* sPtr = (volatile uint32_t*)(shuffleMem + (threadIdx.x& 0xFFFFFFFC));
#else
        volatile uint32_t* sPtr = NULL;
#endif
    if ( thread >= threads )
        return;

    uint32_t tweak1_2[2];
    if (VARIANT > 0) {
        tweak1_2[0] = d_tweak1_2[thread * 2];
        tweak1_2[1] = d_tweak1_2[thread * 2 + 1];
    }

    int i, k;
    uint32_t j;
    const int batchsize = (ITERATIONS * 2) >> ( 2 + bfactor );
    const int start = partidx * batchsize;
    const int end = start + batchsize;
    uint32_t * long_state = &d_long_state[(IndexType) thread << OFFSET];
    uint32_t * ctx_a = d_ctx_a + thread * 4;
    uint32_t * ctx_b = d_ctx_b + thread * 4;
    uint32_t a, d[2];
    uint32_t t1[2], t2[2], res;

    a = ctx_a[sub];
    d[1] = ctx_b[sub];
    #pragma unroll 2
    for ( i = start; i < end; ++i )
    {
        #pragma unroll 2
        for ( int x = 0; x < 2; ++x )
        {
            j = ( ( shuffle(sPtr,sub, a, 0) & MASK) >> 2 ) + sub;

            const uint32_t x_0 = loadGlobal32<uint32_t>( long_state + j );
            const uint32_t x_1 = shuffle(sPtr,sub, x_0, sub + 1);
            const uint32_t x_2 = shuffle(sPtr,sub, x_0, sub + 2);
            const uint32_t x_3 = shuffle(sPtr,sub, x_0, sub + 3);
            d[x] = a ^
                t_fn0( x_0 & 0xff ) ^
                t_fn1( (x_1 >> 8) & 0xff ) ^
                t_fn2( (x_2 >> 16) & 0xff ) ^
                t_fn3( ( x_3 >> 24 ) );


            //XOR_BLOCKS_DST(c, b, &long_state[j]);
            t1[0] = shuffle(sPtr,sub, d[x], 0);
            //long_state[j] = d[0] ^ d[1];

            if (VARIANT > 0) {
                const uint32_t z = d[0] ^ d[1];
                storeGlobal32(long_state + j, sub == 2 ? variant1_1(z) : z);
            }
            else {
                storeGlobal32(long_state + j, d[0] ^ d[1]);
            }

            //MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & MASK]);
            j = ( ( *t1 & MASK) >> 2 ) + sub;

            uint32_t yy[2];
            *( (uint64_t*) yy ) = loadGlobal64<uint64_t>( ( (uint64_t *) long_state )+( j >> 1 ) );
            uint32_t zz[2];
            zz[0] = shuffle(sPtr,sub, yy[0], 0);
            zz[1] = shuffle(sPtr,sub, yy[1], 0);

            t1[1] = shuffle(sPtr,sub, d[x], 1);
            #pragma unroll
            for ( k = 0; k < 2; k++ )
                t2[k] = shuffle(sPtr,sub, a, k + sub2);

            *( (uint64_t *) t2 ) += sub2 ? ( *( (uint64_t *) t1 ) * *( (uint64_t*) zz ) ) : __umul64hi( *( (uint64_t *) t1 ), *( (uint64_t*) zz ) );

            res = *( (uint64_t *) t2 )  >> ( sub & 1 ? 32 : 0 );

            if (VARIANT > 0) {
                storeGlobal32(long_state + j, sub2 ? (tweak1_2[sub & 1] ^ res) : res);
            }
            else {
                storeGlobal32(long_state + j, res);
            }

            a = ( sub & 1 ? yy[1] : yy[0] ) ^ res;
        }
    }

    if ( bfactor > 0 )
    {
        ctx_a[sub] = a;
        ctx_b[sub] = d[1];
    }
}

template<size_t ITERATIONS, size_t OFFSET>
__global__ void cryptonight_core_gpu_phase3( int threads, int bfactor, int partidx, const uint32_t * __restrict__ long_state, uint32_t * __restrict__ d_ctx_state, uint32_t * __restrict__ d_ctx_key2 )
{
    __shared__ uint32_t sharedMemory[1024];

    cn_aes_gpu_init( sharedMemory );
    __syncthreads( );

    int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
    int sub = ( threadIdx.x & 7 ) << 2;

    const int batchsize = ITERATIONS >> bfactor;
    const int start = partidx * batchsize;
    const int end = start + batchsize;

    if ( thread >= threads )
        return;

    uint32_t key[40], text[4];
    MEMCPY8( key, d_ctx_key2 + thread * 40, 20 );
    MEMCPY8( text, d_ctx_state + thread * 50 + sub + 16, 2 );

    __syncthreads( );
    for ( int i = start; i < end; i += 32 )
    {
#pragma unroll
        for ( int j = 0; j < 4; ++j )
            text[j] ^= long_state[((IndexType) thread << OFFSET) + (sub + i + j)];

        cn_aes_pseudo_round_mut( sharedMemory, text, key );
    }

    MEMCPY8( d_ctx_state + thread * 50 + sub + 16, text, 2 );
}


template<size_t ITERATIONS, size_t OFFSET, size_t MASK, uint8_t VARIANT>
void cryptonight_core_cpu_hash(nvid_ctx* ctx)
{
    dim3 grid( ctx->device_blocks );
    dim3 block( ctx->device_threads );
    dim3 block4( ctx->device_threads << 2 );
    dim3 block8( ctx->device_threads << 3 );

    int partcount = 1 << ctx->device_bfactor;

    /* bfactor for phase 1 and 3
     *
     * phase 1 and 3 consume less time than phase 2, therefore we begin with the
     * kernel splitting if the user defined a `bfactor >= 5`
     */
    int bfactorOneThree = ctx->device_bfactor - 4;
    if( bfactorOneThree < 0 )
        bfactorOneThree = 0;

    int partcountOneThree = 1 << bfactorOneThree;

    for ( int i = 0; i < partcountOneThree; i++ )
    {
        CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase1<ITERATIONS, OFFSET><<< grid, block8 >>>(ctx->device_blocks*ctx->device_threads,
            bfactorOneThree, i,
            ctx->d_long_state, ctx->d_ctx_state, ctx->d_ctx_key1));

        if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
    }
    if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );

    for ( int i = 0; i < partcount; i++ )
    {
        CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase2<ITERATIONS, OFFSET, MASK, VARIANT><<<
            grid,
            block4,
            block4.x * sizeof(uint32_t) * static_cast< int >( ctx->device_arch[0] < 3 )
        >>>(
            ctx->device_blocks*ctx->device_threads,
            ctx->device_bfactor,
            i,
            ctx->d_long_state,
            ctx->d_ctx_a,
            ctx->d_ctx_b,
            ctx->d_tweak1_2
        ));

        if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
    }
    for ( int i = 0; i < partcountOneThree; i++ )
    {
        CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase3<ITERATIONS, OFFSET><<< grid, block8 >>>(ctx->device_blocks*ctx->device_threads,
            bfactorOneThree, i,
            ctx->d_long_state,
            ctx->d_ctx_state, ctx->d_ctx_key2));
    }
}


void cryptonight_gpu_hash(nvid_ctx *ctx, int variant, bool lite)
{
#   if !defined(XMRIG_NO_AEON)
    if (lite) {
        if (variant > 0) {
            cryptonight_core_cpu_hash<0x40000, 18, 0x0FFFF0, 1>(ctx);
        }
        else {
            cryptonight_core_cpu_hash<0x40000, 18, 0x0FFFF0, 0>(ctx);
        }

        return;
    }
#   endif

    if (variant > 0) {
        cryptonight_core_cpu_hash<0x80000, 19, 0x1FFFF0, 1>(ctx);
    }
    else {
        cryptonight_core_cpu_hash<0x80000, 19, 0x1FFFF0, 0>(ctx);
    }
}
